
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <cstring>
using namespace std;

__global__ void multiplyDigits(char* d_str1, char* d_str2, int* d_matrix) {
	int row = threadIdx.y;
	int col = threadIdx.x;

	int idx = row * blockDim.x + (col + (blockDim.y * row)) + 1 + (row);

	d_matrix[idx] = (d_str2[row] - '0') * (d_str1[col] - '0');
}

__global__ void propagateCarries(int* d_matrix, int numCols) {
	int idx = threadIdx.x * numCols;
	int carry = 0;

	for (int i = numCols - 1; i >= 0; i--) {
		int rowVal = (d_matrix[idx + i] + carry) % 10;
		carry = (d_matrix[idx + i] + carry) / 10;

		d_matrix[idx + i] = rowVal;
	}
}

__global__ void sumCols(int* d_matrix, int* d_result, int numRows, int numCols) {
	int sum = 0;
	
	for (int i = 0; i < numRows; i++) {
		sum += d_matrix[threadIdx.x + (numCols * i)];
	}

	d_result[threadIdx.x] = sum;
}

__host__ void propagateCarryInFinalResult(int* h_result, int numCols) {
	int carry = 0;

	for (int i = numCols - 1; i >= 0; i--) {
		int rowVal = (h_result[i] + carry) % 10;
		carry = (h_result[i] + carry) / 10;

		h_result[i] = rowVal;
	}
}

int main() {
	char* h_str1 = "123";
	char* h_str2 = "456";

	char* d_str1;
	char* d_str2;

	int* h_matrix;
	int* h_result;

	int* d_matrix;
	int* d_result;

	int row = strlen(h_str2);
	int col = strlen(h_str1) + row;

	h_matrix = new int[row * col];
	h_result = new int[col];

	hipMalloc(&d_str1, sizeof(char) * strlen(h_str1));
	hipMalloc(&d_str2, sizeof(char) * strlen(h_str2));

	hipMalloc(&d_matrix, sizeof(int) * (row * col));
	hipMemset(&d_matrix, 0, sizeof(int) * (row * col));

	hipMalloc(&d_result, sizeof(int) * col);

	hipMemcpy(d_str1, h_str1, sizeof(char) * strlen(h_str1), hipMemcpyHostToDevice);
	hipMemcpy(d_str2, h_str2, sizeof(char) * strlen(h_str2), hipMemcpyHostToDevice);

	multiplyDigits<<<1, dim3(3, 3)>>>(d_str1, d_str2, d_matrix);
	propagateCarries<<<1, 3>>>(d_matrix, col);
	sumCols<<<1, col>>>(d_matrix, d_result, row, col);

	hipMemcpy(h_result, d_result, sizeof(int) * col, hipMemcpyDeviceToHost);

	propagateCarryInFinalResult(h_result, col);

	for (int i = 0; i < col; i++) {
		cout << h_result[i];
	}
	cout << endl;

	hipFree(d_str1);
	hipFree(d_str2);
	hipFree(d_result);

	delete[] h_matrix;
	delete[] h_result;

	cin.get();

	return 0;
}